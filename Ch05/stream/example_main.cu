#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "../include/image_pipeline.h"

// Sobel edge detection kernel
__global__ void sobel_kernel(unsigned char* data, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x <= 0 || x >= width - 1 || y <= 0 || y >= height - 1) return;
    
    // Simple Sobel implementation for grayscale
    int idx = y * width + x;
    
    // Sobel X and Y kernels
    float gx = 0, gy = 0;
    for (int dy = -1; dy <= 1; dy++) {
        for (int dx = -1; dx <= 1; dx++) {
            int sample_idx = (y + dy) * width + (x + dx);
            float val = data[sample_idx];
            
            // Sobel X weights
            if (dx != 0) gx += val * dx * (2 - abs(dy));
            // Sobel Y weights  
            if (dy != 0) gy += val * dy * (2 - abs(dx));
        }
    }
    
    float magnitude = sqrtf(gx * gx + gy * gy);
    data[idx] = fminf(magnitude, 255.0f);
}

// Resize kernel with bilinear interpolation
__global__ void resize_kernel(unsigned char* src, unsigned char* dst,
                              int src_w, int src_h, int dst_w, int dst_h) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= dst_w || y >= dst_h) return;
    
    float src_x = x * (float)(src_w - 1) / (dst_w - 1);
    float src_y = y * (float)(src_h - 1) / (dst_h - 1);
    
    int x0 = floorf(src_x);
    int y0 = floorf(src_y);
    int x1 = min(x0 + 1, src_w - 1);
    int y1 = min(y0 + 1, src_h - 1);
    
    float fx = src_x - x0;
    float fy = src_y - y0;
    
    // Bilinear interpolation
    float p00 = src[y0 * src_w + x0];
    float p01 = src[y0 * src_w + x1];
    float p10 = src[y1 * src_w + x0];
    float p11 = src[y1 * src_w + x1];
    
    float result = (1 - fx) * (1 - fy) * p00 + fx * (1 - fy) * p01 +
                   (1 - fx) * fy * p10 + fx * fy * p11;
    
    dst[y * dst_w + x] = result;
}

// Gaussian blur kernel (simplified)
__global__ void gaussian_blur_kernel(unsigned char* data, int width, int height, float sigma) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    // Simple 3x3 Gaussian kernel
    int kernel_size = 3;
    int half_size = kernel_size / 2;
    
    float sum = 0.0f;
    float weight_sum = 0.0f;
    
    for (int dy = -half_size; dy <= half_size; dy++) {
        for (int dx = -half_size; dx <= half_size; dx++) {
            int sample_x = min(max(x + dx, 0), width - 1);
            int sample_y = min(max(y + dy, 0), height - 1);
            
            float dist_sq = dx * dx + dy * dy;
            float weight = expf(-dist_sq / (2 * sigma * sigma));
            
            sum += data[sample_y * width + sample_x] * weight;
            weight_sum += weight;
        }
    }
    
    // Write to temporary buffer in real implementation
    data[y * width + x] = sum / weight_sum;
}

void launch_sobel(unsigned char* data, int width, int height, hipStream_t stream) {
    dim3 block(16, 16);
    dim3 grid((width + 15) / 16, (height + 15) / 16);
    sobel_kernel<<<grid, block, 0, stream>>>(data, width, height);
}

void launch_resize(unsigned char* src, unsigned char* dst,
                int src_w, int src_h, int dst_w, int dst_h, hipStream_t stream) {
    dim3 block(16, 16);
    dim3 grid((dst_w + 15) / 16, (dst_h + 15) / 16);
    resize_kernel<<<grid, block, 0, stream>>>(src, dst, src_w, src_h, dst_w, dst_h);
}

void launch_gaussian_blur(unsigned char* data, int width, int height,
                        float sigma, hipStream_t stream) {
    dim3 block(16, 16);
    dim3 grid((width + 15) / 16, (height + 15) / 16);
    gaussian_blur_kernel<<<grid, block, 0, stream>>>(data, width, height, sigma);
}


int main() {

    int width = 640;
    int height = 480;
    int channels = 3; 

    uint8_t* h_data = malloc(width * height);
    uint8_t* d_data;
    hipHostAlloc(&d_data, width * height * channels, hipHostMallocDefault);

    for (int i = 0; i < width * height * channels; i++)
    {
        h_data[i] = rand() % 256;
    }
    
    stream_t stream[2];
    for(int i = 0; i < 2; ++i) {
        hipStreamCreate(&stream[i]);
    }

    for(int i = 0; i < 2; i++)
    {
        hipMemcpyAsync(d_data, h_data, width * height * channels, hipMemcpyHostToDevice, stream[i]);
    }

    for (size_t i = 0; i < 2; i++)
    {
        launch_sobel(nullptr, 0, 0, stream[i]);
        // launch_resize(nullptr, nullptr, 0, 0, 0, 0, stream[i]);
        // launch_gaussian_blur(nullptr, 0, 0, 1.0f, stream[i]);
    }

    for (int i = 0; i < 2; ++i) {
        hipStreamSynchronize(stream[i]);
    }

    for (int i = 0; i < 2; ++i) {
        hipMemcpyAsync(h_data, d_data, width * height * channels, hipMemcpyDeviceToHost, stream[i]);
    }

    for(int i = 0; i < 2; ++i) {
        hipStreamDestroy(stream[i]);
    }
    
    hipFree(d_data);
    free(h_data);

    return 0;
}