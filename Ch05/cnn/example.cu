#include <stdio.h>
#include <hip/hip_runtime.h>
#include "include/kernels/activation.h"
#include "include/kernels/convolution.h"


int main() {
    
    int inputSize = 1024;
    size_t bytes = size * sizeof(float);

    float *h_input;
    float *h_output;
    float *h_weight;
    float *h_bias;
    float *d_input;
    float *d_output;
    float *d_weight;
    float *d_bias;

    h_input = (float*)malloc(bytes);
    h_output = (float*)malloc(bytes);
    h_weight = (float*)malloc(bytes);
    h_bias = (float*)malloc(bytes);

    hipMalloc((void**)&d_input, bytes);
    hipMalloc((void**)&d_output, bytes);
    hipMalloc((void**)&d_weight, bytes);
    hipMalloc((void**)&d_bias, bytes);


    // Initialize input, weight, and bias
    for (int i = 0; i < inputSize; i++) {
        h_input[i] = (float)i;
        h_weight[i] = (float)(i + 1);
        h_bias[i] = 0.1f; // Example bias
    }
    
    hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_weight, h_weight, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_bias, h_bias, bytes, hipMemcpyHostToDevice);

    // Define grid and block sizes
    int threadsPerBlock = 256;
    int blocksPerGrid = (inputSize + threadsPerBlock - 1) / threadsPerBlock;

    // Launch convolution kernel
    conv2d_forward<<<blocksPerGrid, threadsPerBlock>>>(
        d_input, d_weight, d_bias, d_output,
        1, 1, 1, 3, 3, 3, 3, 1, 1, 0, 0
    )

    // conv2d_tiled<<<blocksPerGrid, threadsPerBlock, 2 * threadsPerBlock * sizeof(float)>>>(
    //     d_input, d_weight, d_bias, d_output,
    //     1, 1, 1, 3, 3, 3, 3, 1, 1, 0
    // );

    hipDeviceSynchronize();

    relu_forward<<<blocksPerGrid, threadsPerBlock>>>(
        d_output, d_output, inputSize
    );

    hipDeviceSynchronize();

    memcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost);

    free(h_input);
    free(h_output);
    free(h_weight);
    free(h_bias);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_weight);
    hipFree(d_bias);

    
    return 0;
}